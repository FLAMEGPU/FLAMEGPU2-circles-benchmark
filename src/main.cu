#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <chrono>
#include <ctime>

#include "flamegpu/flame_api.h"

// Include the bruteforce implementation

#include "common.cuh"
#include "util.cuh"

// Prototypes for methods from other .cu files
void run_circles_bruteforce(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_bruteforce_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);


bool run_experiment(
    const std::string LABEL,
    const int DEVICE,
    const uint64_t SEED,
    const uint32_t REPETITIONS,
    std::vector<RunSimulationInputs> INPUTS_STRUCTS,
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS
) { 
    printf("Running experiment %s - %zu configs, %zu simulators, %u repetitions\n", LABEL.c_str(), INPUTS_STRUCTS.size(), MODELS.size(), REPETITIONS);

    // Open CSV files
    std::string filename_perSimulationCSV = LABEL + std::string("_perSimulationCSV.csv");
    std::FILE * fp_perSimulationCSV = std::fopen(filename_perSimulationCSV.c_str(), "w");
    if(fp_perSimulationCSV == nullptr) {
        printf("Error: could not open csv file %s\n", filename_perSimulationCSV.c_str());
        std::fclose(fp_perSimulationCSV);
        return false;
    }
    std::string filename_perStepPerSimulationCSV = LABEL + std::string("_perStepPerSimulationCSV.csv");
    std::FILE * fp_perStepPerSimulationCSV = std::fopen(filename_perStepPerSimulationCSV.c_str(), "w");
    if(fp_perStepPerSimulationCSV == nullptr) {
        printf("Error: could not open csv file %s\n", filename_perStepPerSimulationCSV.c_str());
        std::fclose(fp_perSimulationCSV);
        std::fclose(fp_perStepPerSimulationCSV);
        return false;
    }

    // Output the CSV header for each output CSV file.
    if (fp_perSimulationCSV) {
        fprintf(fp_perSimulationCSV, "GPU,release_mode,seatbelts_on,model,steps,agent_count,comm_radius,repeat,mean_messageCount,ms_rtc,ms_simulation,ms_init,ms_exit,ms_step_mean\n");
    }
        
    if (fp_perStepPerSimulationCSV) {
        fprintf(fp_perStepPerSimulationCSV, "GPU,release_mode,seatbelts_on,model,steps,agent_count,comm_radius,repeat,step,ms_step\n");
    }


    const std::string deviceName = getGPUName(DEVICE);
    
    
    const uint32_t totalSimulationCount = INPUTS_STRUCTS.size() * MODELS.size() * REPETITIONS;
    uint32_t simulationIdx = 0; 
    
    // For each input structure
    for (const auto& inputStruct : INPUTS_STRUCTS) {
        for (const auto& modelNameFunctionPair : MODELS) {
            auto const& modelName = modelNameFunctionPair.first;
            auto const& modelFunction = modelNameFunctionPair.second; 
            for (uint32_t repeatIdx = 0u; repeatIdx < REPETITIONS; repeatIdx++){
                // Output progress
                printProgress(modelName, simulationIdx, totalSimulationCount, inputStruct.AGENT_COUNT, inputStruct.COMM_RADIUS, repeatIdx);

                // Run the simulation, capturing values for output.
                const RunSimulationInputs runInputs = {
                    modelName, 
                    inputStruct.HOST_SEED + repeatIdx, // Mutate the seed.
                    inputStruct.AGENT_COUNT, 
                    inputStruct.STEPS, 
                    DEVICE,
                    inputStruct.COMM_RADIUS
                };
                RunSimulationOutputs runOutputs = {};
                modelFunction(runInputs, runOutputs);

                // Add a row to the row per simulation csv file
                if (fp_perSimulationCSV) {
                    fprintf(
                        fp_perSimulationCSV, 
                        "%s,%d,%d,%s,%u,%u,%.3f,%u,%.3f,%.3f,%.3f,%.3f,%.3f,%.3f\n",
                        deviceName.c_str(),
                        isReleaseMode(),
                        isSeatbeltsON(),
                        modelName.c_str(),
                        inputStruct.STEPS,
                        inputStruct.AGENT_COUNT,
                        inputStruct.COMM_RADIUS,
                        repeatIdx,
                        runOutputs.mean_messageCount,
                        runOutputs.ms_rtc,
                        runOutputs.ms_simulation,
                        runOutputs.ms_init,
                        runOutputs.ms_exit,
                        runOutputs.ms_stepMean); 
                }
                // Add a row to the per step per simulation CSV
                if (fp_perStepPerSimulationCSV) {
                    for(uint32_t step = 0; step < runOutputs.ms_per_step->size(); step++){
                        auto& ms_step = runOutputs.ms_per_step->at(step);
                        fprintf(fp_perStepPerSimulationCSV,
                            "%s,%d,%d,%s,%u,%u,%.3f,%u,%u,%.3f\n",
                            deviceName.c_str(),
                            isReleaseMode(),
                            isSeatbeltsON(),
                            modelName.c_str(),
                            inputStruct.STEPS,
                            inputStruct.AGENT_COUNT,
                            inputStruct.COMM_RADIUS,
                            repeatIdx,
                            step,
                            ms_step);
                    }
                }
                simulationIdx++;
            }
        }
    }
    
    // Close csv file handles.
    if(fp_perSimulationCSV){
        std::fclose(fp_perSimulationCSV);
        fp_perSimulationCSV = nullptr; 
    }
    if(fp_perStepPerSimulationCSV) {
        std::fclose(fp_perStepPerSimulationCSV);
        fp_perStepPerSimulationCSV = nullptr; 
    }

    return true;
}


bool experiment_total_scale_all(custom_cli cli){
    // Name the experiment - this will end up in filenames/paths.
    const std::string EXPERIMENT_LABEL="fixed-comm-radius";

    // Select comm radius value(s)
    const float COMM_RADIUS = 2.0f;

    // Select population sizes.
    std::vector<uint32_t> POPULATION_SIZES = {};
    const uint32_t imin = 14u; 
    const uint32_t imax = 15u;
    for(uint32_t i = imin; i < imax; i++){
        POPULATION_SIZES.push_back((1 << i));
        if(i < imax -1){
            POPULATION_SIZES.push_back((1 << i) + (1 << (i-1)));
        }
    }

    // Select the models to execute.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        // {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        // {std::string("circles_bruteforce"), run_circles_bruteforce},
        // {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };

    // Construct the vector of RunSimulationInputs to pass to the run_experiment method.
    auto INPUTS_STRUCTS = std::vector<RunSimulationInputs>();
    for(const auto& popSize : POPULATION_SIZES ){
        INPUTS_STRUCTS.push_back({
            "@todo-modelName", 
            cli.seed,
            popSize, 
            cli.steps, 
            cli.device,
            COMM_RADIUS
        });
    }

    // Run the experriment
    bool success = run_experiment(
        EXPERIMENT_LABEL,
        cli.device,
        cli.seed,
        cli.repetitions,
        INPUTS_STRUCTS,
        MODELS
    );

    return success;
}

bool experiment_density_spatial(const custom_cli cli) {
    // Name the experiment - this will end up in filenames/paths.
    const std::string EXPERIMENT_LABEL="variable-comm-radius";

    // Select comm radius value(s). 2.0f is default. Com radius is related to cuberoot of population....
    std::vector<float> COMM_RADII = {0.25f, 0.5f, 1.0f, 2.0f, 4.0f};

    std::vector<uint32_t> POPULATION_SIZES = {1<<14, 1<<16, 1<<18};
    // std::vector<uint32_t> POPULATION_SIZES = {1<<16};

    // Select population sizes.
    // std::vector<uint32_t> POPULATION_SIZES = {};
    // const uint32_t imin = 14u; 
    // const uint32_t imax = 15u;
    // for(uint32_t i = imin; i < imax; i++){
    //     POPULATION_SIZES.push_back((1 << i));
    //     if(i < imax -1){
    //         POPULATION_SIZES.push_back((1 << i) + (1 << (i-1)));
    //     }
    // }

    // Select the models to execute.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        // {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        // {std::string("circles_bruteforce"), run_circles_bruteforce},
        // {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };

    // Construct the vector of RunSimulationInputs to pass to the run_experiment method.
    auto INPUTS_STRUCTS = std::vector<RunSimulationInputs>();
    for(const auto& popSize : POPULATION_SIZES ){
        for(const auto& commRadius : COMM_RADII) {
            INPUTS_STRUCTS.push_back({
                "@todo-modelName", 
                cli.seed,
                popSize, 
                cli.steps, 
                cli.device,
                commRadius
            });
        }
    }

    // Run the experriment
    bool success = run_experiment(
        EXPERIMENT_LABEL,
        cli.device,
        cli.seed,
        cli.repetitions,
        INPUTS_STRUCTS,
        MODELS
    );

    return success;
}


// @todo - actual device poower state warmup? Maybe run the 0th sim twice and only use the second one?
// @todo deal with what happens if a simulation throws an exception?
int main(int argc, const char ** argv) {
    // Custom arg parsing, to prevent the current F2 arg parsing from occuring. 
    // @todo - improve arg parsing within F2. 
    custom_cli cli = parse_custom_cli(argc, argv);

    // Deal with the CSV output directory and abort if neccesary.
    printf("@todo - output dir via cli (and use it).\n");
    if(false){
        fprintf(stderr, "bad output directory? @todo\n");
        return EXIT_FAILURE;
    }

    // Launch each experiment.
    bool success_1 = experiment_total_scale_all(cli);
    bool success_2 = experiment_density_spatial(cli);

    // exit code
    return success_1 && success_2 ? EXIT_SUCCESS : EXIT_FAILURE;
}


// Todo:

/* 
+ [x] Change the order of loops so pops are first, toa llow early exit.
+ [x] RTC bruteforce
+ [ ] Move pop gen to init fn? so it gets timed.
+ [x] RTC Spatial
+ [x] Better disk io? 
+ [ ] Better error checking. 
+ [x] Plotting (.py)
    + [ ] Headless plotting.
+ [x] density experiment
+ [ ] Individual visualistion
+ [ ] Comments
+ [ ] Seeding?
+ [ ] readme
+ [ ] Check initialisation 
+ [ ] Decide on parameters to use, number of reps
+ [ ] V100 (bessemer) script(s) / trial run. Don't commit these to the public rpo.
+ [ ] limit the scale of some simulators - i.e. bruteforce cpp is horribly slow, so don't push the pops as far. 
+ [x] Have each agent store the message count it read. Exit fn that reduces theses and adds min/max/mean to the output data and CSVs. This might be useful
*/
