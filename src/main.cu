#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <chrono>
#include <ctime>

#include "flamegpu/flame_api.h"

// Include the bruteforce implementation

#include "common.cuh"

void run_circles_bruteforce(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_bruteforce_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);


// Convert some compiler flag values into global constants (if defined) to be output to file
#if defined(NDEBUG) || defined(_NDEBUG)
    const bool RELEASE_MODE = true;
#else 
    const bool RELEASE_MODE = false;
#endif

#if defined(SEATBELTS) && !SEATBELTS
    const bool SEATBELTS_ON = false;
#else 
    const bool SEATBELTS_ON = true;
#endif


void print_cli_help(const int argc, const char ** argv );
custom_cli parse_custom_cli(const int argc, const char ** argv);

void printProgress(const std::string modelName, const uint32_t count, const uint32_t total, const uint32_t agentCount, const uint32_t repeat){
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    char buf[sizeof "2011-10-08T07:07:09Z"];
    std::strftime(buf, sizeof buf, "%FT%TZ", std::gmtime(&now));
    fprintf(stdout, "%s: %u/%u: %s %u %u\n", buf, count + 1, total, modelName.c_str(), agentCount, repeat);
}

// @todo - actual device poower state warmup? Maybe run the 0th sim twice and only use the second one?
// @todo deal with what happens if a simulation throws an exception?
int main(int argc, const char ** argv) {
    // Custom arg parsing, to prevent the current F2 arg parsing from occuring. 
    // @todo - improve arg parsing within F2. 
    custom_cli cli = parse_custom_cli(argc, argv);
    
    // Define the benchmark parameters. I.e. initial environment width, max width, method of interpolation etc. 
    /* std::vector<uint32_t> POPULATION_SIZES = {
        1u << 4,
        1u << 5,
        1u << 6,
        // 1u << 7,
        // 1u << 8,
        // 1u << 9,
        // 1u << 10,
        // 1u << 11,
        // 1u << 12,
        // 1u << 13,
        // 1u << 14,
        // 1u << 15,
        // 1u << 16,
        // 1u << 17,
        // 1u << 18,
        // 1u << 19,
        // 1u << 20,
    }; */


    std::vector<uint32_t> POPULATION_SIZES = {};
    const uint32_t imin = 5; 
    const uint32_t imax = 22;
    for(uint32_t i = imin; i < imax; i++){
        POPULATION_SIZES.push_back((1 << i));
        if(i < imax -1){
            POPULATION_SIZES.push_back((1 << i) + (1 << (i-1)));
        }
    }

    // Define the models to execute, with a function pointer that builds and runs the model.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        {std::string("circles_bruteforce"), run_circles_bruteforce},
        {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };

    // Make the output directory if required.
    printf("@todo - output dir via cli (and use it).\n");

    

    // @todo - error checking, overwrite checking, filename, etc. 
    std::FILE * fp_rowPerSimulation = std::fopen("row-per-simulation.csv", "w");
    if(fp_rowPerSimulation == nullptr){
        printf("could not open file? @todo\n");
        exit(EXIT_FAILURE);
    }
    
    // Output the header for the per run timing.
    fprintf(fp_rowPerSimulation, "GPU,release_mode,seatbelts,model,steps,agentCount,repeat,mean_messageCount,ms_rtc,ms_simulation,ms_init,ms_exit,ms_stepMean\n");
    
    // Write a row per step out to  a differnt file.
    std::FILE * fp_rowPerStepPerSimulation = std::fopen("row-per-step-per-simulation.csv", "w");
    if(fp_rowPerSimulation == nullptr){
        printf("could not open file? @todo\n");
        exit(EXIT_FAILURE);
    }

    // Output a header row. 
    fprintf(fp_rowPerStepPerSimulation, "GPU,release_mode,seatbelts,model,steps,agentCount,repeat,step,ms_step\n");
    
    // Get the name of the gpu. 
    std::string deviceName("unknown");
    hipError_t status = hipSuccess;
    int cudaDeviceCount = 0;
    status = hipGetDeviceCount(&cudaDeviceCount);
    if ( hipSuccess == status) {
        if ( cli.device < cudaDeviceCount ) { 
            hipDeviceProp_t props;
            status = hipGetDeviceProperties(&props, cli.device);
            if (hipSuccess == status) {
                deviceName = std::string(props.name);
                hipFree(0); // Make a context / slightly warm the decice?
            } else {
                printf("@todo handle error \n");
            }
        } else {
            printf("@todo - handle bad cuda device id\n");
        }
    } else {
        printf("@todo handle error \n");
    }
    
    // find the total number of sims to run.
    uint32_t totalSimulations = MODELS.size() * POPULATION_SIZES.size() * cli.repetitions;
    uint32_t counter = 0;

    // Iterate over the population sizes for that model
    for(auto const& agentCount : POPULATION_SIZES){     
        // Iterate the models/simulations to run.
        for(auto const& modelFunctionPair : MODELS){
            auto const& modelName = modelFunctionPair.first;
            auto const& modelFunction = modelFunctionPair.second;    
            // Repeat a number of times to get an average.
            for(uint32_t repeat = 0u; repeat < cli.repetitions; repeat++) { 
                // Progress. 
                printProgress(modelName, counter, totalSimulations, agentCount, repeat);
                // @todo - different seeds? same init differnt device? or both diff?
                const uint64_t seed = cli.seed;
                
                // Run the simulation, capturing values for output.
                const RunSimulationInputs runInputs = {modelName, seed, agentCount, cli.steps, cli.device};
                RunSimulationOutputs runOutputs = {};
                modelFunction(runInputs, runOutputs);

                // Add a row to the row per simulation csv file
                fprintf(fp_rowPerSimulation, "%s,%d,%d,%s,%u,%u,%u,%.3f,%.3f,%.3f,%.3f,%.3f,%.3f\n", deviceName.c_str(), RELEASE_MODE, SEATBELTS_ON, modelName.c_str(), cli.steps, agentCount, repeat, runOutputs.mean_messageCount, runOutputs.ms_rtc, runOutputs.ms_simulation, runOutputs.ms_init, runOutputs.ms_exit, runOutputs.ms_stepMean); 
                
                // add a row to the row per step per simulation file for each step. This is wasting a lot of disk space... 
                for(uint32_t step = 0; step < runOutputs.ms_per_step->size(); step++){
                    auto& ms_step = runOutputs.ms_per_step->at(step);
                    fprintf(fp_rowPerStepPerSimulation, "%s,%d,%d,%s,%u,%u,%u,%u,%.3f\n", deviceName.c_str(), RELEASE_MODE, SEATBELTS_ON, modelName.c_str(), cli.steps, agentCount, repeat, step, ms_step); 
                }
                counter++;
            }
        }
    }
    std::fclose(fp_rowPerStepPerSimulation);
    fp_rowPerStepPerSimulation = nullptr;
    std::fclose(fp_rowPerSimulation);
    fp_rowPerSimulation = nullptr;
}

void print_cli_help(const int argc, const char ** argv ) {
    printf("usage: %s", argv[0]);
    printf(" [-r random]");
    printf(" [-s steps]");
    printf(" [-d device]");
    printf(" [--repetitions repetitions]");
    printf("\n");
    printf("optional args:\n");
    printf("  -r, --random <seed>             Seed for RNG\n");
    printf("  -s, --steps <steps>             Number of simulation iterations\n");
    printf("  -d, --device <device>           CUDA device to use\n");
    printf("      --repetitions <repetitions> The number of benchmark repetitions to perform\n");
}

custom_cli parse_custom_cli(const int argc, const char ** argv) {
    custom_cli values = {};
    // @todo - long term replace this with CLI library which will be included within F2
    for(int i = 0; i < argc; i++){
        std::string arg(argv[i]);
        if(arg.compare("-h") == 0 || arg.compare("--help") == 0){
            print_cli_help(argc, argv);
            exit(EXIT_FAILURE);
        } else if(arg.compare("-r") == 0 || arg.compare("--random-seed") == 0) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.seed = std::stoull(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -r/--random.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -r/--random\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if (arg.compare("-s") == 0 || arg.compare("--steps") == 0 ) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.steps = std::stoul(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -s/--steps argument.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -s/--steps argument\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if (arg.compare("-d") == 0 || arg.compare("--d") == 0 ) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.device = std::stod(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -d/--device argument.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -d/--device argument\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if(arg.compare("--repetitions") == 0) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.repetitions = std::stoul(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for --repetitions.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for --repetitions\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        }
    }
    return values;
}



// Todo:

/* 
+ [x] Change the order of loops so pops are first, toa llow early exit.
+ [x] RTC bruteforce
+ [ ] Move pop gen to init fn? so it gets timed.
+ [x] RTC Spatial
+ [x] Better disk io? 
+ [ ] Better error checking. 
+ [x] Plotting (.py)
    + [ ] Headless plotting.
+ [ ] density experiment
+ [ ] Individual visualistion
+ [ ] Comments
+ [ ] Seeding?
+ [ ] readme
+ [ ] Check initialisation 
+ [ ] Decide on parameters to use, number of reps
+ [ ] V100 (bessemer) script(s) / trial run. Don't commit these to the public rpo.
+ [ ] limit the scale of some simulators - i.e. bruteforce cpp is horribly slow, so don't push the pops as far. 
+ [x] Have each agent store the message count it read. Exit fn that reduces theses and adds min/max/mean to the output data and CSVs. This might be useful
*/
