#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <chrono>
#include <ctime>

#include "flamegpu/flame_api.h"

// Include the bruteforce implementation

#include "common.cuh"

void run_circles_bruteforce(const simMethodParametrs params, simulationTiming &times);
void run_circles_bruteforce_rtc(const simMethodParametrs params, simulationTiming &times);
void run_circles_spatial3D(const simMethodParametrs params, simulationTiming &times);
void run_circles_spatial3D_rtc(const simMethodParametrs params, simulationTiming &times);



// Convert some compiler flag values into global constants (if defined) to be output to file
#if defined(NDEBUG) || defined(_NDEBUG)
    const bool RELEASE_MODE = true;
#else 
    const bool RELEASE_MODE = false;
#endif

#if defined(SEATBELTS) && !SEATBELTS
    const bool SEATBELTS_ON = false;
#else 
    const bool SEATBELTS_ON = true;
#endif


void print_cli_help(const int argc, const char ** argv );
custom_cli parse_custom_cli(const int argc, const char ** argv);
    

__global__ void warmup(float * data, uint32_t elements, uint32_t reps) {

    for(uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < elements; idx += blockDim.x * gridDim.x){
        data[idx] = idx;
        for(uint32_t rep = 0; rep < reps; rep++){
            data[idx] = data[idx] + ((threadIdx.x * rep) % blockDim.x);
        }
    }

}

// Do some arbitrary work on the device to get it  itno a different power state.
void cudaWarmup() {
    // @todo - cuda check.
    const uint32_t elements = 2 << 20; 
    const size_t bytes = elements * sizeof(float);
    const uint32_t reps = 2 << 8; 

    float * d_data = nullptr;
    hipMalloc(&d_data, bytes);
    hipMemset(d_data, 0, bytes);

    int blockSize = 256;
    int gridSize = (elements + blockSize - 1) / blockSize;

    warmup<<<gridSize, blockSize, 0, 0>>>(d_data, elements, reps);
    
    hipFree(d_data);
    d_data = nullptr;
    



}

void printProgress(const std::string modelName, const uint32_t count, const uint32_t total, const uint32_t agentCount, const uint32_t repeat){
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    char buf[sizeof "2011-10-08T07:07:09Z"];
    std::strftime(buf, sizeof buf, "%FT%TZ", std::gmtime(&now));
    fprintf(stdout, "%s: %u/%u: %s %u %u\n", buf, count + 1, total, modelName.c_str(), agentCount, repeat);
}


// @todo deal with what happens if a simulation throws an exception?
int main(int argc, const char ** argv) {
    // Custom arg parsing, to prevent the current F2 arg parsing from occuring. 
    // @todo - improve arg parsing within F2. 
    custom_cli cli = parse_custom_cli(argc, argv);
    
    // Define the benchmark parameters. I.e. initial environment width, max width, method of interpolation etc. 
    // @todo
    std::vector<uint32_t> POPULATION_SIZES = {
        2u << 8,
        2u << 9,
        2u << 10,
        2u << 11,
        2u << 12,
        2u << 13,
        2u << 14,
        2u << 15,
        2u << 16,
        2u << 17,
        2u << 18,
        2u << 19,
        2u << 20,
    };

    // Define the models to execute, with a function pointer that builds and runs the model.
    std::map<std::string, std::function<void(const simMethodParametrs, simulationTiming&)>> MODELS = {
        // {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        // {std::string("circles_bruteforce"), run_circles_bruteforce},
        {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };


    // Make the output directory if required.
    printf("@todo - output dir via cli (and use it).\n");

    
    // Write out the parameters used to generate the model? This might not be required as it can be figured out from the actual data...

    // std::ofstream paramsFile("params.csv");
    // @todo - output directory.
    // @todo - add cuda device name. 
    // if (paramsFile.is_open()) {
    //     paramsFile << initialPopSize << "," << finalPopSize << "," << popSizeIncrement << std::endl;
    //     paramsFile << initialNumSpecies << "," << finalNumSpecies << "," << numSpeciesIncrement << std::endl;
    // }

    // @todo - error checking, overwrite checking, filename, etc. 
    std::FILE * fp_rowPerSimulation = std::fopen("row-per-simulation.csv", "w");
    if(fp_rowPerSimulation == nullptr){
        printf("could not open file? @todo\n");
        exit(EXIT_FAILURE);
    }
    
    
    // Output the header for the per run timing.
    fprintf(fp_rowPerSimulation, "GPU, release_mode, seatbelts, model, steps, agentCount, repeat, ms_rtc, ms_simulation, ms_init, ms_exit, ms_stepMean\n");
    

    
    // Get the name of the gpu. 
    std::string deviceName("unknown");
    hipError_t status = hipSuccess;
    int cudaDeviceCount = 0;
    status = hipGetDeviceCount(&cudaDeviceCount);
    if ( hipSuccess == status) {
        if ( cli.device < cudaDeviceCount ) { 
            hipDeviceProp_t props;
            status = hipGetDeviceProperties(&props, cli.device);
            if (hipSuccess == status) {
                deviceName = std::string(props.name);
                hipFree(0); // Make a context / slightly warm the decice?
                cudaWarmup();
            } else {
                printf("@todo handle error \n");
            }
        } else {
            printf("@todo - handle bad cuda device id\n");
        }
    } else {
        printf("@todo handle error \n");
    }
    
    // find the total number of sims to run.
    uint32_t totalSimulations = MODELS.size() * POPULATION_SIZES.size() * cli.repetitions;
    uint32_t counter = 0;

    // Iterate over population size first. This then allows for early exit when sims become too slow? Alternatively do the fastest simulations first, but this would require changing the map to be ordered.
    // Iterate the models/simulations to run.
    for(auto const& modelFunctionPair : MODELS){
        auto const& modelName = modelFunctionPair.first;
        auto const& modelFunction = modelFunctionPair.second;    
        
        // @todo - do (some) aggregation of timers? e.g. output 1 csv for a given sim-popsize combo, with step times for each sim and mean step times? for 
        
        // Iterate over the population sizes for that model
        for(auto const& agentCount : POPULATION_SIZES){            
            // Repeat a number of times to get an average.
            for(uint32_t repeat = 0u; repeat < cli.repetitions; repeat++) { 

                // Progress. 
                printProgress(modelName, counter, totalSimulations, agentCount, repeat);

                // @todo - better filenaming
                std::string simulationStepsRawFilename = std::string("simulation-steps-raw-") + std::to_string(counter) + std::string(".csv");
                // @todo better file handling, directory etc.
                std::FILE * fp_simulationStepsRaw = std::fopen(simulationStepsRawFilename.c_str(), "w");
                if(fp_simulationStepsRaw == nullptr){
                    printf("could not open file? @todo\n");
                    exit(EXIT_FAILURE);
                }
                // @todo - Do i need to use a different seed for each repetition? Probably should do both...
                const uint64_t seed = cli.seed;
                
                // Call the fn to run this simulation witht his pop for this rep. 
                // @todo get timing info to save for alter.
                simulationTiming t = {};
                modelFunction({modelName, seed, agentCount, cli.steps, cli.device}, t);


                // Output the individual runs times
                fprintf(fp_rowPerSimulation, "\"%s\", %d, %d, \"%s\", %u, %u, %u, %.3f, %.3f, %.3f, %.3f, %.3f\n", deviceName.c_str(), RELEASE_MODE, SEATBELTS_ON, modelName.c_str(), cli.steps, agentCount, repeat, t.ms_rtc, t.ms_simulation, t.ms_init, t.ms_exit, t.ms_stepMean); 
                
                // Output a csv containing the per step time 
                fprintf(fp_simulationStepsRaw, "GPU, release_mode, seatbelts, model, steps, agentCount, repeat, step, ms_step\n");
                for(uint32_t step = 0; step < t.ms_per_step->size(); step++){
                    auto& ms_step = t.ms_per_step->at(step);
                    fprintf(fp_simulationStepsRaw, "\"%s\", %d, %d, \"%s\", %u, %u, %u, %u, %.3f\n", deviceName.c_str(), RELEASE_MODE, SEATBELTS_ON, modelName.c_str(), cli.steps, agentCount, repeat, step, ms_step); 
                }

                std::fclose(fp_simulationStepsRaw);
                fp_simulationStepsRaw = nullptr;

                counter++;
            }
        }
    }
    std::fclose(fp_rowPerSimulation);
    fp_rowPerSimulation = nullptr;
}




void print_cli_help(const int argc, const char ** argv ) {
    printf("usage: %s", argv[0]);
    printf(" [-r random]");
    printf(" [-s steps]");
    printf(" [-d device]");
    printf(" [--repetitions repetitions]");
    printf("\n");
    printf("optional args:\n");
    printf("  -r, --random <seed>             Seed for RNG\n");
    printf("  -s, --steps <steps>             Number of simulation iterations\n");
    printf("  -d, --device <device>           CUDA device to use\n");
    printf("      --repetitions <repetitions> The number of benchmark repetitions to perform\n");
}

custom_cli parse_custom_cli(const int argc, const char ** argv) {
    custom_cli values = {};
    // @todo - long term replace this with CLI library which will be included within F2
    for(int i = 0; i < argc; i++){
        std::string arg(argv[i]);
        if(arg.compare("-h") == 0 || arg.compare("--help") == 0){
            print_cli_help(argc, argv);
            exit(EXIT_FAILURE);
        } else if(arg.compare("-r") == 0 || arg.compare("--random-seed") == 0) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.seed = std::stoull(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -r/--random.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -r/--random\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if (arg.compare("-s") == 0 || arg.compare("--steps") == 0 ) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.steps = std::stoul(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -s/--steps argument.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -s/--steps argument\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if (arg.compare("-d") == 0 || arg.compare("--d") == 0 ) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.device = std::stod(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -d/--device argument.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -d/--device argument\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if(arg.compare("--repetitions") == 0) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.repetitions = std::stoul(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for --repetitions.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for --repetitions\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        }
    }

    return values;
}



// Todo:

/* 
+ [ ] Change the order of loops so pops are first, toa llow early exit.
+ [ ] RTC bruteforce
+ [ ] Move pop gen to init fn? so it gets timed.
+ [ ] RTC Spatial
+ [ ] Better disk io? 
    + [ ] Combine the per-step time files somehow? Maybe even just cat them into a very tall, repettitive csv?
+ [ ] Better error checking. 
+ [ ] Plotting (.py)
    + [ ] Headless plotting.
+ [ ] density experiment
+ [ ] Individual visualistion
+ [ ] Comments
+ [ ] Seeding?
+ [ ] readme
+ [ ] Check initialisation 
+ [ ] Decide on parameters to use, number of reps
+ [ ] V100 (bessemer) script(s) / trial run. Don't commit these to the public rpo.
+ [ ] limit the scale of some simulators - i.e. bruteforce cpp is horribly slow, so don't push the pops as far. 
+ [ ] Have each agent store the message count it read. Exit fn that reduces theses and adds min/max/mean to the output data and CSVs. This might be useful
*/
