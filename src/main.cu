#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <chrono>
#include <ctime>

#include "flamegpu/flame_api.h"

// Include the bruteforce implementation

#include "common.cuh"

void run_circles_bruteforce(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_bruteforce_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);


// Convert some compiler flag values into global constants (if defined) to be output to file
#if defined(NDEBUG) || defined(_NDEBUG)
    const bool RELEASE_MODE = true;
#else 
    const bool RELEASE_MODE = false;
#endif

#if defined(SEATBELTS) && !SEATBELTS
    const bool SEATBELTS_ON = false;
#else 
    const bool SEATBELTS_ON = true;
#endif


void print_cli_help(const int argc, const char ** argv );
custom_cli parse_custom_cli(const int argc, const char ** argv);

void printProgress(const std::string modelName, const uint32_t count, const uint32_t total, const uint32_t agentCount, const uint32_t repeat){
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    char buf[sizeof "2011-10-08T07:07:09Z"];
    std::strftime(buf, sizeof buf, "%FT%TZ", std::gmtime(&now));
    fprintf(stdout, "%s: %u/%u: %s %u %u\n", buf, count + 1, total, modelName.c_str(), agentCount, repeat);
}

// @todo - actual device poower state warmup? Maybe run the 0th sim twice and only use the second one?
// @todo deal with what happens if a simulation throws an exception?
int main(int argc, const char ** argv) {
    // Custom arg parsing, to prevent the current F2 arg parsing from occuring. 
    // @todo - improve arg parsing within F2. 
    custom_cli cli = parse_custom_cli(argc, argv);
    
    // Define the benchmark parameters. I.e. initial environment width, max width, method of interpolation etc. 
    // @todo
    std::vector<uint32_t> POPULATION_SIZES = {
        2u << 8,
        2u << 9,
        // 2u << 10,
        // 2u << 11,
        // 2u << 12,
        // 2u << 13,
        // 2u << 14,
        // 2u << 15,
        // 2u << 16,
        // 2u << 17,
        // 2u << 18,
        // 2u << 19,
        // 2u << 20,
    };

    // Define the models to execute, with a function pointer that builds and runs the model.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        {std::string("circles_bruteforce"), run_circles_bruteforce},
        {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };

    // Make the output directory if required.
    printf("@todo - output dir via cli (and use it).\n");

    

    // @todo - error checking, overwrite checking, filename, etc. 
    std::FILE * fp_rowPerSimulation = std::fopen("row-per-simulation.csv", "w");
    if(fp_rowPerSimulation == nullptr){
        printf("could not open file? @todo\n");
        exit(EXIT_FAILURE);
    }
    
    // Output the header for the per run timing.
    fprintf(fp_rowPerSimulation, "GPU,release_mode,seatbelts,model,steps,agentCount,repeat,ms_rtc,ms_simulation,ms_init,ms_exit,ms_stepMean\n");
    
    /* std::string s = std::string("");
    for(uint32_t step = 0; step < cli.steps; step++){
        s += std::string(",ms_step_") + std::to_string(step);
    }
    fprintf(fp_rowPerSimulation, "%s\n", s.c_str()); */


    // Write a row per step out to  a differnt file.
    std::FILE * fp_rowPerStepPerSimulation = std::fopen("row-per-step-per-simulation.csv", "w");
    if(fp_rowPerSimulation == nullptr){
        printf("could not open file? @todo\n");
        exit(EXIT_FAILURE);
    }

    // Output a header row. 
    fprintf(fp_rowPerStepPerSimulation, "GPU,release_mode,seatbelts,model,steps,agentCount,repeat,step,ms_step\n");
    
    // Get the name of the gpu. 
    std::string deviceName("unknown");
    hipError_t status = hipSuccess;
    int cudaDeviceCount = 0;
    status = hipGetDeviceCount(&cudaDeviceCount);
    if ( hipSuccess == status) {
        if ( cli.device < cudaDeviceCount ) { 
            hipDeviceProp_t props;
            status = hipGetDeviceProperties(&props, cli.device);
            if (hipSuccess == status) {
                deviceName = std::string(props.name);
                hipFree(0); // Make a context / slightly warm the decice?
            } else {
                printf("@todo handle error \n");
            }
        } else {
            printf("@todo - handle bad cuda device id\n");
        }
    } else {
        printf("@todo handle error \n");
    }
    
    // find the total number of sims to run.
    uint32_t totalSimulations = MODELS.size() * POPULATION_SIZES.size() * cli.repetitions;
    uint32_t counter = 0;

    // Iterate over population size first. This then allows for early exit when sims become too slow? Alternatively do the fastest simulations first, but this would require changing the map to be ordered.
    // Iterate the models/simulations to run.
    for(auto const& modelFunctionPair : MODELS){
        auto const& modelName = modelFunctionPair.first;
        auto const& modelFunction = modelFunctionPair.second;    
        
        // @todo - do (some) aggregation of timers? e.g. output 1 csv for a given sim-popsize combo, with step times for each sim and mean step times? for 
        
        // Iterate over the population sizes for that model
        for(auto const& agentCount : POPULATION_SIZES){     
            // Repeat a number of times to get an average.
            for(uint32_t repeat = 0u; repeat < cli.repetitions; repeat++) { 

                // Progress. 
                printProgress(modelName, counter, totalSimulations, agentCount, repeat);
                // @todo - Do i need to use a different seed for each repetition? Probably should do both...
                const uint64_t seed = cli.seed;
                
                // Call the fn to run this simulation witht his pop for this rep. 
                // @todo get timing info to save for alter.
                const RunSimulationInputs runInputs = {modelName, seed, agentCount, cli.steps, cli.device};
                RunSimulationOutputs runOutputs = {};
                modelFunction(runInputs, runOutputs);


                // Add a row to the row per simulation csv file
                fprintf(fp_rowPerSimulation, "%s,%d,%d,%s,%u,%u,%u,%.3f,%.3f,%.3f,%.3f,%.3f\n", deviceName.c_str(), RELEASE_MODE, SEATBELTS_ON, modelName.c_str(), cli.steps, agentCount, repeat, runOutputs.ms_rtc, runOutputs.ms_simulation, runOutputs.ms_init, runOutputs.ms_exit, runOutputs.ms_stepMean); 
                
                // add a row to the row per step per simulation file for each step. This is wasting a lot of disk space... 
                for(uint32_t step = 0; step < runOutputs.ms_per_step->size(); step++){
                    auto& ms_step = runOutputs.ms_per_step->at(step);
                    fprintf(fp_rowPerStepPerSimulation, "%s,%d,%d,%s,%u,%u,%u,%u,%.3f\n", deviceName.c_str(), RELEASE_MODE, SEATBELTS_ON, modelName.c_str(), cli.steps, agentCount, repeat, step, ms_step); 
                }
                counter++;
            }
        }
    }
    std::fclose(fp_rowPerStepPerSimulation);
    fp_rowPerStepPerSimulation = nullptr;
    std::fclose(fp_rowPerSimulation);
    fp_rowPerSimulation = nullptr;
}




void print_cli_help(const int argc, const char ** argv ) {
    printf("usage: %s", argv[0]);
    printf(" [-r random]");
    printf(" [-s steps]");
    printf(" [-d device]");
    printf(" [--repetitions repetitions]");
    printf("\n");
    printf("optional args:\n");
    printf("  -r, --random <seed>             Seed for RNG\n");
    printf("  -s, --steps <steps>             Number of simulation iterations\n");
    printf("  -d, --device <device>           CUDA device to use\n");
    printf("      --repetitions <repetitions> The number of benchmark repetitions to perform\n");
}

custom_cli parse_custom_cli(const int argc, const char ** argv) {
    custom_cli values = {};
    // @todo - long term replace this with CLI library which will be included within F2
    for(int i = 0; i < argc; i++){
        std::string arg(argv[i]);
        if(arg.compare("-h") == 0 || arg.compare("--help") == 0){
            print_cli_help(argc, argv);
            exit(EXIT_FAILURE);
        } else if(arg.compare("-r") == 0 || arg.compare("--random-seed") == 0) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.seed = std::stoull(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -r/--random.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -r/--random\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if (arg.compare("-s") == 0 || arg.compare("--steps") == 0 ) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.steps = std::stoul(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -s/--steps argument.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -s/--steps argument\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if (arg.compare("-d") == 0 || arg.compare("--d") == 0 ) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.device = std::stod(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for -d/--device argument.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for -d/--device argument\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        } else if(arg.compare("--repetitions") == 0) {
            if(i + 1 < argc){
                try {
                    std::string v(argv[i+1]);
                    values.repetitions = std::stoul(v);
                } catch (const std::exception& e){
                    printf("Error: Invalid value for --repetitions.");
                    exit(EXIT_FAILURE);
                }
            } else {
                printf("Error: Missing value for --repetitions\n");
                print_cli_help(argc, argv);
                exit(EXIT_FAILURE);
            }
        }
    }

    return values;
}



// Todo:

/* 
+ [ ] Change the order of loops so pops are first, toa llow early exit.
+ [x] RTC bruteforce
+ [ ] Move pop gen to init fn? so it gets timed.
+ [x] RTC Spatial
+ [ ] Better disk io? 
    + [ ] Combine the per-step time files somehow? Maybe even just cat them into a very tall, repettitive csv?
+ [ ] Better error checking. 
+ [ ] Plotting (.py)
    + [ ] Headless plotting.
+ [ ] density experiment
+ [ ] Individual visualistion
+ [ ] Comments
+ [ ] Seeding?
+ [ ] readme
+ [ ] Check initialisation 
+ [ ] Decide on parameters to use, number of reps
+ [ ] V100 (bessemer) script(s) / trial run. Don't commit these to the public rpo.
+ [ ] limit the scale of some simulators - i.e. bruteforce cpp is horribly slow, so don't push the pops as far. 
+ [ ] Have each agent store the message count it read. Exit fn that reduces theses and adds min/max/mean to the output data and CSVs. This might be useful
*/
