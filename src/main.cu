#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <chrono>
#include <ctime>

#include "flamegpu/flamegpu.h"
#include "common.cuh"
#include "util.cuh"

#define DRY_RUN 0

// Prototypes for methods from other .cu files
void run_circles_bruteforce(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_bruteforce_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_bruteforce_sorted(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_bruteforce_rtc_sorted(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);


bool run_experiment(
    const std::string LABEL,
    const int DEVICE,
    const uint32_t REPETITIONS,
    std::vector<RunSimulationInputs> INPUTS_STRUCTS,
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS,
    const bool dry 
) { 
    printf("Running experiment %s - %zu configs, %zu simulators, %u repetitions\n", LABEL.c_str(), INPUTS_STRUCTS.size(), MODELS.size(), REPETITIONS);

    // Open CSV files
    std::string filename_perSimulationCSV = LABEL + std::string("_perSimulationCSV.csv");
    std::FILE * fp_perSimulationCSV = std::fopen(filename_perSimulationCSV.c_str(), "w");
    if(fp_perSimulationCSV == nullptr) {
        printf("Error: could not open csv file %s\n", filename_perSimulationCSV.c_str());
        std::fclose(fp_perSimulationCSV);
        return false;
    }
    std::string filename_perStepPerSimulationCSV = LABEL + std::string("_perStepPerSimulationCSV.csv");
    std::FILE * fp_perStepPerSimulationCSV = std::fopen(filename_perStepPerSimulationCSV.c_str(), "w");
    if(fp_perStepPerSimulationCSV == nullptr) {
        printf("Error: could not open csv file %s\n", filename_perStepPerSimulationCSV.c_str());
        std::fclose(fp_perSimulationCSV);
        std::fclose(fp_perStepPerSimulationCSV);
        return false;
    }

    // Output the CSV header for each output CSV file.
    if (fp_perSimulationCSV) {
        fprintf(fp_perSimulationCSV, "GPU,release_mode,seatbelts_on,model,steps,agent_count,env_width,comm_radius,sort_period,repeat,agent_density,mean_message_count,s_rtc,s_simulation,s_init,s_exit,s_step_mean\n");
    }
        
    if (fp_perStepPerSimulationCSV) {
        fprintf(fp_perStepPerSimulationCSV, "GPU,release_mode,seatbelts_on,model,steps,agent_count,env_width,comm_radius,sort_period,repeat,agent_density,step,s_step\n");
    }


    const std::string deviceName = getGPUName(DEVICE);
    
    
    const uint32_t totalSimulationCount = INPUTS_STRUCTS.size() * MODELS.size() * REPETITIONS;
    uint32_t simulationIdx = 0; 
    
    // For each input structure
    for (const auto& inputStruct : INPUTS_STRUCTS) {
        for (const auto& modelNameFunctionPair : MODELS) {
            auto const& modelName = modelNameFunctionPair.first;
            auto const& modelFunction = modelNameFunctionPair.second; 
            for (uint32_t repeatIdx = 0u; repeatIdx < REPETITIONS; repeatIdx++){
                // Output progress
                printProgress(
                    modelName, 
                    simulationIdx, 
                    totalSimulationCount, 
                    inputStruct.AGENT_COUNT, 
                    inputStruct.ENV_WIDTH, 
                    inputStruct.COMM_RADIUS, 
                    repeatIdx);

                // Only print the progress if a dry run.
                if (dry) { 
                    continue;
                }

                // Run the simulation, capturing values for output.
                const RunSimulationInputs runInputs = {
                    DEVICE,
                    inputStruct.STEPS, 
                    inputStruct.SEED + repeatIdx,
                    inputStruct.AGENT_COUNT, 
                    inputStruct.ENV_WIDTH,
                    inputStruct.COMM_RADIUS,
                    inputStruct.SORT_PERIOD
                };
                RunSimulationOutputs runOutputs = {};
                modelFunction(runInputs, runOutputs);

                // Add a row to the row per simulation csv file
                if (fp_perSimulationCSV) {
                    fprintf(
                        fp_perSimulationCSV, 
                        "%s,%d,%d,%s,%u,%u,%.6f,%.6f,%u,%u,%.6f,%.6f,%.6f,%.6f,%.6f,%.6f,%.6f\n",
                        deviceName.c_str(),
                        isReleaseMode(),
                        isSeatbeltsON(),
                        modelName.c_str(),
                        inputStruct.STEPS,
                        inputStruct.AGENT_COUNT,
                        inputStruct.ENV_WIDTH,
                        inputStruct.COMM_RADIUS,
                        inputStruct.SORT_PERIOD,
                        repeatIdx,
                        runOutputs.agentDensity,
                        runOutputs.mean_messageCount,
                        runOutputs.s_rtc,
                        runOutputs.s_simulation,
                        runOutputs.s_init,
                        runOutputs.s_exit,
                        runOutputs.s_stepMean); 
                }
                // Add a row to the per step per simulation CSV
                if (fp_perStepPerSimulationCSV) {
                    for(uint32_t step = 0; step < runOutputs.s_per_step->size(); step++){
                        auto& s_step = runOutputs.s_per_step->at(step);
                        fprintf(fp_perStepPerSimulationCSV,
                            "%s,%d,%d,%s,%u,%u,%.6f,%.6f,%u,%u,%.6f,%u,%.6f\n",
                            deviceName.c_str(),
                            isReleaseMode(),
                            isSeatbeltsON(),
                            modelName.c_str(),
                            inputStruct.STEPS,
                            inputStruct.AGENT_COUNT,
                            inputStruct.ENV_WIDTH,
                            inputStruct.COMM_RADIUS,
                            inputStruct.SORT_PERIOD,
                            repeatIdx,
                            runOutputs.agentDensity,
                            step,
                            s_step);
                    }
                }
                simulationIdx++;
            }
        }
    }
    
    // Close csv file handles.
    if(fp_perSimulationCSV){
        std::fclose(fp_perSimulationCSV);
        fp_perSimulationCSV = nullptr; 
    }
    if(fp_perStepPerSimulationCSV) {
        std::fclose(fp_perStepPerSimulationCSV);
        fp_perStepPerSimulationCSV = nullptr; 
    }

    return true;
}


bool experiment_total_scale_all(custom_cli cli){
    // Name the experiment - this will end up in filenames/paths.
    const std::string EXPERIMENT_LABEL="fixed-density";

    // Fixed comm radius 
    const float COMM_RADIUS = 2.f;
    // Fixed density
    const float DENSITY = 1.0f;
    // Fixed sort period
    const uint32_t SORT_PERIOD = 1u; 

    // Sweep over environment widths, which lead to scaled 
    // Env width needs to be atleast 5 * comm_radius to not read all messages? (so that there are bins in atleast each dim?)
    // @density 1, 8 width = 512 pop. 16 = 4k, 20 = 8k, 40 width = 64k pop, 100 = 1million.
    // const std::vector<float> ENV_WIDTHS = {8.f, 12.f, 16.f, 20.f};
    // const std::vector<float> ENV_WIDTHS = {8.f, 12.f, 16.f, 20.f, 30.f, 40.f, 50.f, 60.f, 70.f, 80.f, 90.f, 100.f};
    std::vector<float> ENV_WIDTHS = {};


    const std::vector<float> TARGET_ENV_VOLUMES = {10000, 20000, 30000, 40000, 50000, 60000, 70000, 80000, 90000, 100000, 200000, 300000, 400000, 500000, 600000, 700000, 800000, 900000, 1000000};
    for(const float& targetVolume : TARGET_ENV_VOLUMES){
        const float envWidth = round(cbrt(targetVolume));
        const float actualVolume = envWidth * envWidth * envWidth;
        const float badness = (actualVolume - targetVolume) / targetVolume;
        ENV_WIDTHS.push_back(envWidth);
        // printf("targetVolume %f actualVolume %f width %f, volumeBadness %f\n", targetVolume, actualVolume, envWidth, badness);
    }


    // Select the models to execute.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        {std::string("circles_bruteforce"), run_circles_bruteforce},
        {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };

    // Construct the vector of RunSimulationInputs to pass to the run_experiment method.
    auto INPUTS_STRUCTS = std::vector<RunSimulationInputs>();
    // for(const auto& popSize : POPULATION_SIZES ){
        // const float envWidth = static_cast<float>(ceil(cbrt(popSize)));
    for(const auto& envWidth : ENV_WIDTHS ){
        const uint32_t popSize = static_cast<float>(ceil((envWidth * envWidth * envWidth) * DENSITY)); 
        // Envwidth is scaled with population size.
        INPUTS_STRUCTS.push_back({
            cli.device,
            cli.steps,
            cli.seed,
            popSize,
            envWidth,
            COMM_RADIUS,
            SORT_PERIOD
        });
    }

    // Run the experriment
    bool success = run_experiment(
        EXPERIMENT_LABEL,
        cli.device,
        cli.repetitions,
        INPUTS_STRUCTS,
        MODELS,
        cli.dry
    );

    return success;
}

bool experiment_density_spatial(const custom_cli cli) {
    // Name the experiment - this will end up in filenames/paths.
    const std::string EXPERIMENT_LABEL="variable-density";

    // Vary the density / environment width for several agent populations.
    std::vector<float> COMM_VOLUME_FRACTIONS = {};

    // Fixed comm radius 
    const float COMM_RADIUS = 2.f;

    const uint32_t SORT_PERIOD = 1u; 
    // Sweep over densities.
    // std::vector<float> DENSITIES = {1.f, 2.f, 4.f, 8.f}; 
    // std::vector<float> DENSITIES = {1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 8.f, 9.f, 10.f}; 
    
    // Sweep over environment widths, which lead to scaled 
    // std::vector<float> ENV_WIDTHS = {8.f, 20.f, 40.f};
    // const std::vector<float> ENV_WIDTHS = {8.f, 12.f, 16.f, 20.f, 30.f, 40.f, 50.f, 60.f, 70.f, 80.f, 90.f, 100.f};
    // const std::vector<float> ENV_WIDTHS = {40, 50, 60, 70};

    std::vector<float> DENSITIES = {1.f, 2.f, 3.f, 4.f}; 
    const std::vector<float> TARGET_ENV_VOLUMES = {10000, 20000, 30000, 40000, 50000, 60000, 70000, 80000, 90000, 100000, 150000, 200000, 250000, 300000,  350000, 400000, 450000, 500000};
    std::vector<float> ENV_WIDTHS = {};
    for(const float& targetVolume : TARGET_ENV_VOLUMES){
        const float envWidth = round(cbrt(targetVolume));
        const float actualVolume = envWidth * envWidth * envWidth;
        const float badness = (actualVolume - targetVolume) / targetVolume;
        ENV_WIDTHS.push_back(envWidth);
        // printf("targetVolume %f actualVolume %f width %f, volumeBadness %f\n", targetVolume, actualVolume, envWidth, badness);
    }



    // Select the models to execute.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        // {std::string("circles_bruteforce"), run_circles_bruteforce},
        // {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };

    // Construct the vector of RunSimulationInputs to pass to the run_experiment method.
    auto INPUTS_STRUCTS = std::vector<RunSimulationInputs>();
    for(const auto& envWidth : ENV_WIDTHS ){
        for(const auto& density : DENSITIES ){
            const uint32_t popSize = static_cast<float>(ceil((envWidth * envWidth * envWidth) * density)); 
            // Envwidth is scaled with population size.
            INPUTS_STRUCTS.push_back({
                cli.device,
                cli.steps,
                cli.seed,
                popSize,
                envWidth,
                COMM_RADIUS,
                SORT_PERIOD
            });
        }
    }

    // Run the experriment
    bool success = run_experiment(
        EXPERIMENT_LABEL,
        cli.device,
        cli.repetitions,
        INPUTS_STRUCTS,
        MODELS,
        cli.dry
    );

    return success;
}

bool experiment_sort_period(custom_cli cli){
    // Name the experiment - this will end up in filenames/paths.
    const std::string EXPERIMENT_LABEL="sort-period";
    
    const uint32_t popSize = 64000;
    const float ENV_WIDTH = 40.0f;  

    const std::vector<float> comm_radii = {2.0f, 4.0f, 6.0f, 8.0f};
    const std::vector<uint32_t> sortPeriods = {0u, 1u, 2u, 5u, 10u, 20u, 50u, 100u, 200u}; 

    // Select the models to execute.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        //{std::string("circles_bruteforce"), run_circles_bruteforce},
        //{std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
        //{std::string("circles_bruteforce_sorted"), run_circles_bruteforce_sorted},
        //{std::string("circles_bruteforce_rtc_sorted"), run_circles_bruteforce_rtc_sorted},
    };

    // Construct the vector of RunSimulationInputs to pass to the run_experiment method.
    auto INPUTS_STRUCTS = std::vector<RunSimulationInputs>();
    for(const auto& sortPeriod : sortPeriods){
	for(const auto& comm_radius : comm_radii) {
            // Envwidth is scaled with population size.
            INPUTS_STRUCTS.push_back({
                cli.device,
                cli.steps,
                cli.seed,
                popSize,
                ENV_WIDTH,
                comm_radius,
                sortPeriod
            });
	}
    }

    // Run the experriment
    bool success = run_experiment(
        EXPERIMENT_LABEL,
        cli.device,
        cli.repetitions,
        INPUTS_STRUCTS,
        MODELS,
        cli.dry
    );

    return success;
}

bool experiment_comm_radius(custom_cli cli){
    // Name the experiment - this will end up in filenames/paths.
    const std::string EXPERIMENT_LABEL="comm-radius";
    
    const uint32_t popSize = 64000;
    const float ENV_WIDTH = 40.0f;  
    const uint32_t SORT_PERIOD = 1u; 
    const std::vector<float> comm_radii = {2.0f, 4.0f, 6.0f, 8.0f, 10.0f, 12.0f, 14.0f, 16.0f, 18.0f, 20.0f, 22.0f, 24.0f, 26.0f, 28.0f, 30.0f, 32.0f, 34.0f, 36.0f, 38.0f, 40.0f};

    // Select the models to execute.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        {std::string("circles_bruteforce"), run_circles_bruteforce},
        {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
        {std::string("circles_bruteforce_sorted"), run_circles_bruteforce_sorted},
        {std::string("circles_bruteforce_rtc_sorted"), run_circles_bruteforce_rtc_sorted},
    };

    // Construct the vector of RunSimulationInputs to pass to the run_experiment method.
    auto INPUTS_STRUCTS = std::vector<RunSimulationInputs>();
    for(const auto& comm_radius : comm_radii ){
        // Envwidth is scaled with population size.
        INPUTS_STRUCTS.push_back({
            cli.device,
            cli.steps,
            cli.seed,
            popSize,
            ENV_WIDTH,
            comm_radius,
            SORT_PERIOD
        });
    }

    // Run the experriment
    bool success = run_experiment(
        EXPERIMENT_LABEL,
        cli.device,
        cli.repetitions,
        INPUTS_STRUCTS,
        MODELS,
        cli.dry
    );

    return success;
}

int main(int argc, const char ** argv) {
    // Custom arg parsing, to prevent the current F2 arg parsing from occuring. 
    custom_cli cli = parse_custom_cli(argc, argv);

    // Launch each experiment.
    //bool success_1 = experiment_total_scale_all(cli);
    //bool success_2 = experiment_density_spatial(cli);
    //bool success_3 = experiment_comm_radius(cli);
    bool success_4 = experiment_sort_period(cli);

    // exit code
    //return success_1 && success_2 && success_3 ? EXIT_SUCCESS : EXIT_FAILURE;
    return success_4 ? EXIT_SUCCESS : EXIT_FAILURE;
}
