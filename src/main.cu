#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <chrono>
#include <ctime>

#include "flamegpu/flame_api.h"
#include "common.cuh"
#include "util.cuh"

#define SEED_PRIME 97
#define DRY_RUN 0

// Prototypes for methods from other .cu files
void run_circles_bruteforce(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_bruteforce_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);
void run_circles_spatial3D_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs);


bool run_experiment(
    const std::string LABEL,
    const int DEVICE,
    const uint64_t SEED,
    const uint32_t REPETITIONS,
    std::vector<RunSimulationInputs> INPUTS_STRUCTS,
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS,
    const bool dry 
) { 
    printf("Running experiment %s - %zu configs, %zu simulators, %u repetitions\n", LABEL.c_str(), INPUTS_STRUCTS.size(), MODELS.size(), REPETITIONS);

    // Open CSV files
    std::string filename_perSimulationCSV = LABEL + std::string("_perSimulationCSV.csv");
    std::FILE * fp_perSimulationCSV = std::fopen(filename_perSimulationCSV.c_str(), "w");
    if(fp_perSimulationCSV == nullptr) {
        printf("Error: could not open csv file %s\n", filename_perSimulationCSV.c_str());
        std::fclose(fp_perSimulationCSV);
        return false;
    }
    std::string filename_perStepPerSimulationCSV = LABEL + std::string("_perStepPerSimulationCSV.csv");
    std::FILE * fp_perStepPerSimulationCSV = std::fopen(filename_perStepPerSimulationCSV.c_str(), "w");
    if(fp_perStepPerSimulationCSV == nullptr) {
        printf("Error: could not open csv file %s\n", filename_perStepPerSimulationCSV.c_str());
        std::fclose(fp_perSimulationCSV);
        std::fclose(fp_perStepPerSimulationCSV);
        return false;
    }

    // Output the CSV header for each output CSV file.
    if (fp_perSimulationCSV) {
        fprintf(fp_perSimulationCSV, "GPU,release_mode,seatbelts_on,model,steps,agent_count,env_width,comm_radius,repeat,agent_density,mean_message_count,ms_rtc,ms_simulation,ms_init,ms_exit,ms_step_mean,pre_flame_used_bytes,pre_flame_free_bytes,flame_used_bytes,flame_free_bytes\n");
    }
        
    if (fp_perStepPerSimulationCSV) {
        fprintf(fp_perStepPerSimulationCSV, "GPU,release_mode,seatbelts_on,model,steps,agent_count,env_width,comm_radius,repeat,agent_density,step,ms_step\n");
    }


    const std::string deviceName = getGPUName(DEVICE);
    
    
    const uint32_t totalSimulationCount = INPUTS_STRUCTS.size() * MODELS.size() * REPETITIONS;
    uint32_t simulationIdx = 0; 
    
    // For each input structure
    for (const auto& inputStruct : INPUTS_STRUCTS) {
        for (const auto& modelNameFunctionPair : MODELS) {
            auto const& modelName = modelNameFunctionPair.first;
            auto const& modelFunction = modelNameFunctionPair.second; 
            for (uint32_t repeatIdx = 0u; repeatIdx < REPETITIONS; repeatIdx++){
                // Output progress
                printProgress(
                    modelName, 
                    simulationIdx, 
                    totalSimulationCount, 
                    inputStruct.AGENT_COUNT, 
                    inputStruct.ENV_WIDTH, 
                    inputStruct.COMM_RADIUS, 
                    repeatIdx);

                // Only print the progress if a dry run.
                if (dry) { 
                    continue;
                }

                // Run the simulation, capturing values for output.
                const RunSimulationInputs runInputs = {
                    DEVICE,
                    inputStruct.STEPS, 
                    inputStruct.HOST_SEED + (repeatIdx * SEED_PRIME), // Mutate the seed.
                    inputStruct.AGENT_COUNT, 
                    inputStruct.ENV_WIDTH,
                    inputStruct.COMM_RADIUS
                };
                RunSimulationOutputs runOutputs = {};
                modelFunction(runInputs, runOutputs);

                // If the run was successful, output csv, otherwise skip.
                // @todo - ideally this would also not attempt to run any larger sims, but oh well.
                if(runOutputs.completed) {
                    // Add a row to the row per simulation csv file
                    if (fp_perSimulationCSV) {
                        fprintf(
                            fp_perSimulationCSV, 
                            "%s,%d,%d,%s,%u,%u,%.3f,%.3f,%u,%.3f,%.3f,%.3f,%.3f,%.3f,%.3f,%.3f,%zu,%zu,%zu,%zu\n",
                            deviceName.c_str(),
                            isReleaseMode(),
                            isSeatbeltsON(),
                            modelName.c_str(),
                            inputStruct.STEPS,
                            inputStruct.AGENT_COUNT,
                            inputStruct.ENV_WIDTH,
                            inputStruct.COMM_RADIUS,
                            repeatIdx,
                            runOutputs.agentDensity,
                            runOutputs.mean_messageCount,
                            runOutputs.ms_rtc,
                            runOutputs.ms_simulation,
                            runOutputs.ms_init,
                            runOutputs.ms_exit,
                            runOutputs.ms_stepMean,
                            runOutputs.preFlameUsedBytes,
                            runOutputs.preFlameFreeBytes,
                            runOutputs.flameUsedBytes,
                            runOutputs.flameFreeBytes); 
                        fflush(fp_perSimulationCSV);
                    }
                    // Add a row to the per step per simulation CSV
                    if (fp_perStepPerSimulationCSV) {
                        for(uint32_t step = 0; step < runOutputs.ms_per_step->size(); step++){
                            auto& ms_step = runOutputs.ms_per_step->at(step);
                            fprintf(fp_perStepPerSimulationCSV,
                                "%s,%d,%d,%s,%u,%u,%.3f,%.3f,%u,%.3f,%u,%.3f\n",
                                deviceName.c_str(),
                                isReleaseMode(),
                                isSeatbeltsON(),
                                modelName.c_str(),
                                inputStruct.STEPS,
                                inputStruct.AGENT_COUNT,
                                inputStruct.ENV_WIDTH,
                                inputStruct.COMM_RADIUS,
                                repeatIdx,
                                runOutputs.agentDensity,
                                step,
                                ms_step);
                            fflush(fp_perStepPerSimulationCSV);
                        }
                    }
                }
                simulationIdx++;
            }
        }
    }
    
    // Close csv file handles.
    if(fp_perSimulationCSV){
        std::fclose(fp_perSimulationCSV);
        fp_perSimulationCSV = nullptr; 
    }
    if(fp_perStepPerSimulationCSV) {
        std::fclose(fp_perStepPerSimulationCSV);
        fp_perStepPerSimulationCSV = nullptr; 
    }

    return true;
}


bool experiment_total_scale_all(custom_cli cli){
    // Name the experiment - this will end up in filenames/paths.
    const std::string EXPERIMENT_LABEL="fixed-density";

    // Fixed comm radius 
    const float COMM_RADIUS = 2.f;
    // Fixed density
    const float DENSITY = 1.0f; 

    // Sweep over environment widths, which lead to scaled 
    // Env width needs to be atleast 5 * comm_radius to not read all messages? (so that there are bins in atleast each dim?)
    // @density 1, 8 width = 512 pop. 16 = 4k, 20 = 8k, 40 width = 64k pop, 100 = 1million.
    // const std::vector<float> ENV_WIDTHS = {8.f, 12.f, 16.f, 20.f};
    // const std::vector<float> ENV_WIDTHS = {8.f, 12.f, 16.f, 20.f, 30.f, 40.f, 50.f, 60.f, 70.f, 80.f, 90.f, 100.f};
    std::vector<float> ENV_WIDTHS = {};

    // @todo - need to try catch calls to simulate() if I want to sample multiple repetitions, and output some kind of error value (and then abort any larger sizes.)

    // const std::vector<float> TARGET_ENV_VOLUMES = {10000, 20000, 30000, 40000, 50000, 60000, 70000, 80000, 90000, 100000, 200000, 300000, 400000, 500000, 600000, 700000, 800000, 900000, 1000000};

    // Start with some small target volumes to get some sampling at the low end.
    std::vector<float> TARGET_ENV_VOLUMES = {1, 32, 128, 256, 512, 1024, 4096, 16000, 32000, 64000};
    // Then sample at every 100k upto 1 million.
    for(uint32_t i = 1; i < 10; i++){
        const float scaleFactor = 100000.f;
        TARGET_ENV_VOLUMES.push_back(i * scaleFactor);
    }
    // Then sample at every 1million upto 10 milliion 
    for(uint32_t i = 1; i < 10; i++){
        const float scaleFactor = 1000000.f;
        TARGET_ENV_VOLUMES.push_back(i * scaleFactor);
    }
    // Then sample at every 10 million upto 200 million.
    for(uint32_t i = 1; i < 21; i++){
        const float scaleFactor = 10000000.f;
        TARGET_ENV_VOLUMES.push_back(i * scaleFactor);
    }

    for(const float& targetVolume : TARGET_ENV_VOLUMES){
        const float envWidth = round(cbrt(targetVolume));
        const float actualVolume = envWidth * envWidth * envWidth;
        const float badness = (actualVolume - targetVolume) / targetVolume;
        ENV_WIDTHS.push_back(envWidth);
        // printf("targetVolume %f actualVolume %f width %f, volumeBadness %f\n", targetVolume, actualVolume, envWidth, badness);
    }
    // exit(1);


    // Select the models to execute.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        // {std::string("circles_spatial3D"), run_circles_spatial3D},
        {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        // {std::string("circles_bruteforce"), run_circles_bruteforce},
        {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };

    // Construct the vector of RunSimulationInputs to pass to the run_experiment method.
    auto INPUTS_STRUCTS = std::vector<RunSimulationInputs>();
    // for(const auto& popSize : POPULATION_SIZES ){
        // const float envWidth = static_cast<float>(ceil(cbrt(popSize)));
    for(const auto& envWidth : ENV_WIDTHS ){
        const uint32_t popSize = static_cast<float>(ceil((envWidth * envWidth * envWidth) * DENSITY)); 
        // Envwidth is scaled with population size.
        INPUTS_STRUCTS.push_back({
            cli.device,
            cli.steps,
            cli.seed,
            popSize,
            envWidth,
            COMM_RADIUS
        });
    }

    // Run the experriment
    bool success = run_experiment(
        EXPERIMENT_LABEL,
        cli.device,
        cli.seed,
        cli.repetitions,
        INPUTS_STRUCTS,
        MODELS,
        cli.dry
    );

    return success;
}

bool experiment_density_spatial(const custom_cli cli) {
    // Name the experiment - this will end up in filenames/paths.
    const std::string EXPERIMENT_LABEL="variable-density";

    // Vary the density / environment width for several agent populations.
    std::vector<float> COMM_VOLUME_FRACTIONS = {};

    // Fixed comm radius 
    const float COMM_RADIUS = 2.f;

    // Sweep over densities.
    // std::vector<float> DENSITIES = {1.f, 2.f, 4.f, 8.f}; 
    // std::vector<float> DENSITIES = {1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 8.f, 9.f, 10.f}; 
    
    // Sweep over environment widths, which lead to scaled 
    // std::vector<float> ENV_WIDTHS = {8.f, 20.f, 40.f};
    // const std::vector<float> ENV_WIDTHS = {8.f, 12.f, 16.f, 20.f, 30.f, 40.f, 50.f, 60.f, 70.f, 80.f, 90.f, 100.f};
    // const std::vector<float> ENV_WIDTHS = {40, 50, 60, 70};

    std::vector<float> DENSITIES = {1.f, 2.f, 3.f, 4.f}; 
    const std::vector<float> TARGET_ENV_VOLUMES = {10000, 20000, 30000, 40000, 50000, 60000, 70000, 80000, 90000, 100000, 150000, 200000, 250000, 300000,  350000, 400000, 450000, 500000};
    std::vector<float> ENV_WIDTHS = {};
    for(const float& targetVolume : TARGET_ENV_VOLUMES){
        const float envWidth = round(cbrt(targetVolume));
        const float actualVolume = envWidth * envWidth * envWidth;
        const float badness = (actualVolume - targetVolume) / targetVolume;
        ENV_WIDTHS.push_back(envWidth);
        // printf("targetVolume %f actualVolume %f width %f, volumeBadness %f\n", targetVolume, actualVolume, envWidth, badness);
    }



    // Select the models to execute.
    std::map<std::string, std::function<void(const RunSimulationInputs, RunSimulationOutputs&)>> MODELS = {
        // {std::string("circles_spatial3D"), run_circles_spatial3D},
        // {std::string("circles_spatial3D_rtc"), run_circles_spatial3D_rtc},
        // {std::string("circles_bruteforce"), run_circles_bruteforce},
        // {std::string("circles_bruteforce_rtc"), run_circles_bruteforce_rtc},
    };

    // Construct the vector of RunSimulationInputs to pass to the run_experiment method.
    auto INPUTS_STRUCTS = std::vector<RunSimulationInputs>();
    for(const auto& envWidth : ENV_WIDTHS ){
        for(const auto& density : DENSITIES ){
            const uint32_t popSize = static_cast<float>(ceil((envWidth * envWidth * envWidth) * density)); 
            // Envwidth is scaled with population size.
            INPUTS_STRUCTS.push_back({
                cli.device,
                cli.steps,
                cli.seed,
                popSize,
                envWidth,
                COMM_RADIUS
            });
        }
    }

    // Run the experriment
    bool success = run_experiment(
        EXPERIMENT_LABEL,
        cli.device,
        cli.seed,
        cli.repetitions,
        INPUTS_STRUCTS,
        MODELS,
        cli.dry
    );

    return success;
}


int main(int argc, const char ** argv) {
    // Custom arg parsing, to prevent the current F2 arg parsing from occuring. 
    // @todo - improve arg parsing within F2. 
    custom_cli cli = parse_custom_cli(argc, argv);

    // Deal with the CSV output directory and abort if neccesary.
    printf("@todo - output dir via cli (and use it).\n");
    if(false){
        fprintf(stderr, "bad output directory? @todo\n");
        return EXIT_FAILURE;
    }

    // Launch each experiment.
    bool success_1 = experiment_total_scale_all(cli);
    bool success_2 = experiment_density_spatial(cli);

    // exit code
    return success_1 && success_2 ? EXIT_SUCCESS : EXIT_FAILURE;
}

/* 
// Todo:
+ [x] Change the order of loops so pops are first, toa llow early exit.
+ [x] RTC bruteforce
+ [ ] Move pop gen to init fn? so it gets timed? Agent vec in init would be good.
+ [x] RTC Spatial
+ [ ] Output to a specified directory (which may or may not exist?)
+ [ ] Better error checking. 
    + [ ] if a simulation throws an exception?
    + [ ] If could not create the csv file
+ [x] Plotting (.py)
+ [x] Headless plotting.
+ [x] density experiment
+ [ ] Individual visualistion
+ [ ] Comments
+ [x] Seeding?
+ [ ] readme
+ [ ] Check initialisation 
+ [ ] Decide on parameters to use, number of reps
+ [x] V100 (bessemer) script(s) / trial run. Don't commit these to the public rpo.
+ [ ] limit the scale of some simulators - i.e. bruteforce cpp is horribly slow, so don't push the pops as far. 
+ [x] Have each agent store the message count it read. Exit fn that reduces theses and adds min/max/mean to the output data and CSVs. This might be useful
+ [ ]actual device poower state warmup? Maybe run the 0th sim twice and only use the second one?
*/