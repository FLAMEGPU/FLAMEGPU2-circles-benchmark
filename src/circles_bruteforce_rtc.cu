#include "hip/hip_runtime.h"
#include <algorithm>

#include "flamegpu/flame_api.h"
#include "common.cuh"

namespace {
const char * output_message = R"###(
FLAMEGPU_AGENT_FUNCTION(output_message, MsgNone, MsgBruteForce) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
    return ALIVE;
}
)###";

const char * move = R"###(
FLAMEGPU_AGENT_FUNCTION(move, MsgBruteForce, MsgNone) {
    const int ID = FLAMEGPU->getVariable<int>("id");
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->environment.getProperty<float>("radius");
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in) {
        if (message.getVariable<int>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = cbrt(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", cbrt(fx*fx + fy*fy + fz*fz));
    return ALIVE;
}
)###";

#if defined(CIRCLES_VALIDATION) && CIRCLES_VALIDATION
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
#endif 
}  // namespace

// Run an individual simulation, using 
void run_circles_bruteforce_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs){
    ModelDescription model("circles_bruteforce_rtc");
    // Calculate environment bounds.
    const float ENV_WIDTH = runInputs.ENV_WIDTH;
    const float ENV_MIN = -0.5 * ENV_WIDTH;
    const float ENV_MAX = ENV_MIN + ENV_WIDTH;
    // Compute the actual density and return it.
    runOutputs.agentDensity = runInputs.AGENT_COUNT / (ENV_WIDTH * ENV_WIDTH * ENV_WIDTH);

    {   // Location message
        MsgBruteForce::Description &message = model.newMessage<MsgBruteForce>("location");
        message.newVariable<int>("id");
        message.newVariable<float>("x");
        message.newVariable<float>("y");
        message.newVariable<float>("z");
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("Circle");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newRTCFunction("output_message", output_message).setMessageOutput("location");
        agent.newRTCFunction("move", move).setMessageInput("location");
    }

    // Global environment variables.
    {
        EnvironmentDescription &env = model.Environment();
        env.newProperty("repulse", ENV_REPULSE);
        env.newProperty("radius", runInputs.COMM_RADIUS);
    }

    // Organise the model. 

#if defined(CIRCLES_VALIDATION) && CIRCLES_VALIDATION
    {   // Attach init/step/exit functions and exit condition
        model.addStepFunction(Validation);
    }
#endif  // CIRCLES_VALIDATION

    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction("Circle", "output_message");
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction("Circle", "move");
    }

    // Create the simulation object
    CUDASimulation simulation(model);

    // Set config configuraiton properties 
    simulation.SimulationConfig().timing = false;
    simulation.SimulationConfig().verbose = false;
    simulation.SimulationConfig().random_seed = runInputs.HOST_SEED;  // @todo device seed != host seed? 
    simulation.SimulationConfig().steps = runInputs.STEPS;
    simulation.CUDAConfig().device_id = runInputs.CUDA_DEVICE;

    // Generate the initial population
    std::default_random_engine rng(runInputs.HOST_SEED);
    std::uniform_real_distribution<float> dist(ENV_MIN, ENV_MAX);
    AgentVector population(model.Agent("Circle"), runInputs.AGENT_COUNT);
    for (unsigned int i = 0; i < runInputs.AGENT_COUNT; i++) {
        AgentVector::Agent instance = population[i];
        instance.setVariable<int>("id", i);
        instance.setVariable<float>("x", dist(rng));
        instance.setVariable<float>("y", dist(rng));
        instance.setVariable<float>("z", dist(rng));
    }

    // Set the population for the simulation.
    simulation.setPopulationData(population);

    // Execute 
    simulation.simulate();

    // Store timing information for later use.
    runOutputs.ms_rtc = simulation.getElapsedTimeRTCInitialisation();
    runOutputs.ms_simulation = simulation.getElapsedTimeSimulation();
    runOutputs.ms_init = simulation.getElapsedTimeInitFunctions();
    runOutputs.ms_exit = simulation.getElapsedTimeExitFunctions();
    
    std::vector<float> ms_steps = simulation.getElapsedTimeSteps();
    runOutputs.ms_per_step = std::make_shared<std::vector<float>>(std::vector<float>(ms_steps.begin(), ms_steps.end()));
    runOutputs.ms_stepMean = std::accumulate(ms_steps.begin(), ms_steps.end(), 0.f) / (float)simulation.getStepCounter();
    runOutputs.mean_messageCount = runInputs.AGENT_COUNT;
}