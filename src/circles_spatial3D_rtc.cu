#include "hip/hip_runtime.h"
#include <algorithm>

#include "flamegpu/flame_api.h"
#include "common.cuh"

namespace {

const char * output_message = R"###(
FLAMEGPU_AGENT_FUNCTION(output_message, MsgNone, MsgSpatial3D) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"),
        FLAMEGPU->getVariable<float>("z"));
    return ALIVE;
}
)###";

const char * move = R"###(
FLAMEGPU_AGENT_FUNCTION(move, MsgSpatial3D, MsgNone) {
    const int ID = FLAMEGPU->getVariable<int>("id");
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->message_in.radius();
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    int messageCount = 0;
    for (const auto &message : FLAMEGPU->message_in(x1, y1, z1)) {
        if (message.getVariable<int>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = cbrt(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
        messageCount++;
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", cbrt(fx*fx + fy*fy + fz*fz));
    float totalMessageCount = FLAMEGPU->getVariable<float>("totalMessageCount");
    FLAMEGPU->setVariable<float>("totalMessageCount", totalMessageCount + messageCount);
    return ALIVE;
}
)###";

static float meanMessageCount = 0.f;
static size_t preFlameUsedBytes = 0u;
static size_t preFlameFreeBytes = 0u;
static size_t flameUsedBytes = 0u;
static size_t flameFreeBytes = 0u;

FLAMEGPU_EXIT_FUNCTION(getMeanMessageCount) {
    
    const float totalMessageCount = FLAMEGPU->agent("Circle").sum<float>("totalMessageCount") / FLAMEGPU->agent("Circle").count();
    
    meanMessageCount = totalMessageCount / FLAMEGPU->getStepCounter();
    
    size_t totalBytes = 0u;
    static size_t usedBytes = 0u;
    hipMemGetInfo(&flameFreeBytes, &totalBytes);
    usedBytes = totalBytes - flameFreeBytes;
    flameUsedBytes = usedBytes - preFlameUsedBytes;
    // printf("Exit free %zu, total %zu, used %zu, flame used %zu\n", flameFreeBytes, totalBytes, usedBytes, flameUsedBytes);

}

#if defined(CIRCLES_VALIDATION) && CIRCLES_VALIDATION
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
#endif
}  // namespace

// Run an individual simulation, using 
void run_circles_spatial3D_rtc(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs){
    size_t totalBytes = 0u;
    hipMemGetInfo(&preFlameFreeBytes, &totalBytes);
    preFlameUsedBytes = totalBytes - preFlameFreeBytes;
    // printf("Init free %zu, total %zu used %zu\n", preFlameFreeBytes, totalBytes, preFlameUsedBytes);
    
    ModelDescription model("circles_spatial3D_rtc");
    // Calculate environment bounds.
    const float ENV_WIDTH = runInputs.ENV_WIDTH;
    const float ENV_MIN = -0.5 * ENV_WIDTH;
    const float ENV_MAX = ENV_MIN + ENV_WIDTH;
    // Compute the actual density and return it.
    runOutputs.agentDensity = runInputs.AGENT_COUNT / (ENV_WIDTH * ENV_WIDTH * ENV_WIDTH);

    {   // Location message
        MsgSpatial3D::Description &message = model.newMessage<MsgSpatial3D>("location");
        message.newVariable<int>("id");
        message.setRadius(runInputs.COMM_RADIUS);
        message.setMin(ENV_MIN, ENV_MIN, ENV_MIN);
        message.setMax(ENV_MAX, ENV_MAX, ENV_MAX);
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("Circle");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("totalMessageCount", 0.f);
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newRTCFunction("output_message", output_message).setMessageOutput("location");
        agent.newRTCFunction("move", move).setMessageInput("location");
    }

    // Global environment variables.
    {
        EnvironmentDescription &env = model.Environment();
        env.newProperty("repulse", ENV_REPULSE);
    }

    // Organise the model. 

    model.addExitFunction(getMeanMessageCount);


#if defined(CIRCLES_VALIDATION) && CIRCLES_VALIDATION
    {   // Attach init/step/exit functions and exit condition
        model.addStepFunction(Validation);
    }
#endif  // CIRCLES_VALIDATION

    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction("Circle", "output_message");
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction("Circle", "move");
    }

    // Create the simulation object
    CUDASimulation simulation(model);

    // Set config configuraiton properties 
    simulation.SimulationConfig().timing = false;
    simulation.SimulationConfig().verbose = false;
    simulation.SimulationConfig().random_seed = runInputs.HOST_SEED;  // @todo device seed != host seed? 
    simulation.SimulationConfig().steps = runInputs.STEPS;
    simulation.CUDAConfig().device_id = runInputs.CUDA_DEVICE;

    // Generate the initial population
    std::default_random_engine rng(runInputs.HOST_SEED);
    std::uniform_real_distribution<float> dist(ENV_MIN, ENV_MAX);
    AgentVector population(model.Agent("Circle"), runInputs.AGENT_COUNT);
    for (unsigned int i = 0; i < runInputs.AGENT_COUNT; i++) {
        AgentVector::Agent instance = population[i];
        instance.setVariable<int>("id", i);
        instance.setVariable<float>("x", dist(rng));
        instance.setVariable<float>("y", dist(rng));
        instance.setVariable<float>("z", dist(rng));
    }

    // Set the population for the simulation.
    simulation.setPopulationData(population);

    // Execute 
    simulation.simulate();

    // Store timing information for later use.
    runOutputs.ms_rtc = simulation.getElapsedTimeRTCInitialisation();
    runOutputs.ms_simulation = simulation.getElapsedTimeSimulation();
    runOutputs.ms_init = simulation.getElapsedTimeInitFunctions();
    runOutputs.ms_exit = simulation.getElapsedTimeExitFunctions();
    
    std::vector<float> ms_steps = simulation.getElapsedTimeSteps();
    runOutputs.ms_per_step = std::make_shared<std::vector<float>>(std::vector<float>(ms_steps.begin(), ms_steps.end()));
    runOutputs.ms_stepMean = std::accumulate(ms_steps.begin(), ms_steps.end(), 0.f) / (float)simulation.getStepCounter();
    runOutputs.mean_messageCount = meanMessageCount;

    runOutputs.preFlameUsedBytes = preFlameUsedBytes;
    runOutputs.preFlameFreeBytes = preFlameFreeBytes;
    runOutputs.flameUsedBytes = flameUsedBytes;
    runOutputs.flameFreeBytes = flameFreeBytes;
}