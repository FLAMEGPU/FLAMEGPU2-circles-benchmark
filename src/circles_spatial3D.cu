#include "hip/hip_runtime.h"
#include <algorithm>

#include "flamegpu/flame_api.h"
#include "common.cuh"

namespace {

FLAMEGPU_AGENT_FUNCTION(output_message, MsgNone, MsgSpatial3D) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setLocation(
    FLAMEGPU->getVariable<float>("x"),
    FLAMEGPU->getVariable<float>("y"),
    FLAMEGPU->getVariable<float>("z"));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(move, MsgSpatial3D, MsgNone) {
    const int ID = FLAMEGPU->getVariable<int>("id");
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->message_in.radius();
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in(x1, y1, z1)) {
        if (message.getVariable<int>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = cbrt(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", cbrt(fx*fx + fy*fy + fz*fz));
    return ALIVE;
}

#if defined(CIRCLES_VALIDATION) && CIRCLES_VALIDATION
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
#endif

}  // namespace

// @todo - ad a way to visualise a single run of a single simulator somehow? maybe -v/--visualise <model_name> <pop>

// Run an individual simulation, using 
void run_circles_spatial3D(const simMethodParametrs params, simulationTiming &times){

    ModelDescription model(params.modelName);
    const float ENV_MAX = static_cast<float>(floor(cbrt(params.AGENT_COUNT)));
    {   // Location message
        MsgSpatial3D::Description &message = model.newMessage<MsgSpatial3D>("location");
        message.newVariable<int>("id");
        message.setRadius(COMM_RADIUS);
        message.setMin(0, 0, 0);
        message.setMax(ENV_MAX, ENV_MAX, ENV_MAX);
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("Circle");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newFunction("output_message", output_message).setMessageOutput("location");
        agent.newFunction("move", move).setMessageInput("location");
    }

    // Global environment variables.
    {
        EnvironmentDescription &env = model.Environment();
        env.newProperty("repulse", ENV_REPULSE);
    }

    // Organise the model. 

#if defined(CIRCLES_VALIDATION) && CIRCLES_VALIDATION
    {   // Attach init/step/exit functions and exit condition
        model.addStepFunction(Validation);
    }
#endif  // CIRCLES_VALIDATION

    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(output_message);
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(move);
    }

    // Create the simulation object
    CUDASimulation simulation(model);

    // Set config configuraiton properties 
    simulation.SimulationConfig().timing = false;
    simulation.SimulationConfig().verbose = false;
    simulation.SimulationConfig().random_seed = params.HOST_SEED;  // @todo device seed != host seed? 
    simulation.SimulationConfig().steps = params.STEPS;
    simulation.CUDAConfig().device_id = params.CUDA_DEVICE;

    // Generate the initial population
    std::default_random_engine rng(params.HOST_SEED);
    std::uniform_real_distribution<float> dist(0.0f, ENV_MAX);
    AgentVector population(model.Agent("Circle"), params.AGENT_COUNT);
    for (unsigned int i = 0; i < params.AGENT_COUNT; i++) {
        AgentVector::Agent instance = population[i];
        instance.setVariable<int>("id", i);
        instance.setVariable<float>("x", dist(rng));
        instance.setVariable<float>("y", dist(rng));
        instance.setVariable<float>("z", dist(rng));
    }

    // Set the population for the simulation.
    simulation.setPopulationData(population);

    // Execute 
    simulation.simulate();

    // Store timing information for later use.
    times.ms_rtc = simulation.getElapsedTimeRTCInitialisation();
    times.ms_simulation = simulation.getElapsedTimeSimulation();
    times.ms_init = simulation.getElapsedTimeInitFunctions();
    times.ms_exit = simulation.getElapsedTimeExitFunctions();
    
    std::vector<float> ms_steps = simulation.getElapsedTimeSteps();
    times.ms_per_step = std::make_shared<std::vector<float>>(std::vector<float>(ms_steps.begin(), ms_steps.end()));
    times.ms_stepMean = std::accumulate(ms_steps.begin(), ms_steps.end(), 0.f) / (float)simulation.getStepCounter();
}