#include "hip/hip_runtime.h"
#include <algorithm>

#include "flamegpu/flamegpu.h"
#include "common.cuh"

namespace {

const char * output_message = R"###(
FLAMEGPU_AGENT_FUNCTION(output_message, flamegpu::MessageNone, flamegpu::MessageBruteForceSorted) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"),
        FLAMEGPU->getVariable<float>("z"));
    return flamegpu::ALIVE;
}
)###";

const char * move = R"###(
FLAMEGPU_AGENT_FUNCTION(move, flamegpu::MessageBruteForceSorted, flamegpu::MessageNone) {
    const int ID = FLAMEGPU->getVariable<int>("id");
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->message_in.radius();
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    int messageCount = 0;
    for (const auto &message : FLAMEGPU->message_in) {
        if (message.getVariable<int>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = cbrt(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
        messageCount++;
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", cbrt(fx*fx + fy*fy + fz*fz));
    float totalMessageCount = FLAMEGPU->getVariable<float>("totalMessageCount");
    FLAMEGPU->setVariable<float>("totalMessageCount", totalMessageCount + messageCount);
    return flamegpu::ALIVE;
}
)###";

static float meanMessageCount = 0.f;
FLAMEGPU_EXIT_FUNCTION(getMeanMessageCount) {

    const float totalMessageCount = FLAMEGPU->agent("Circle").sum<float>("totalMessageCount") / FLAMEGPU->agent("Circle").count();

    meanMessageCount = totalMessageCount / FLAMEGPU->getStepCounter();
}

#if defined(CIRCLES_VALIDATION) && CIRCLES_VALIDATION
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
#endif
}  // namespace

// Run an individual simulation, using 
void run_circles_bruteforce_rtc_sorted(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs){
    flamegpu::ModelDescription model("circles_bruteforce_rtc_sorted");
    // Calculate environment bounds.
    const float ENV_WIDTH = runInputs.ENV_WIDTH;
    const float ENV_MIN = -0.5 * ENV_WIDTH;
    const float ENV_MAX = ENV_MIN + ENV_WIDTH;
    // Compute the actual density and return it.
    runOutputs.agentDensity = runInputs.AGENT_COUNT / (ENV_WIDTH * ENV_WIDTH * ENV_WIDTH);

    {   // Location message
        flamegpu::MessageBruteForceSorted::Description &message = model.newMessage<flamegpu::MessageBruteForceSorted>("location");
        message.newVariable<int>("id");
        message.setRadius(runInputs.COMM_RADIUS);
        message.setMin(ENV_MIN, ENV_MIN, ENV_MIN);
        message.setMax(ENV_MAX, ENV_MAX, ENV_MAX);
    }
    {   // Circle agent
        flamegpu::AgentDescription &agent = model.newAgent("Circle");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("totalMessageCount", 0.f);
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newRTCFunction("output_message", output_message).setMessageOutput("location");
        agent.newRTCFunction("move", move).setMessageInput("location");
        agent.setSortPeriod(runInputs.SORT_PERIOD);
    }

    // Global environment variables.
    {
        flamegpu::EnvironmentDescription &env = model.Environment();
        env.newProperty("repulse", ENV_REPULSE);
    }

    // Organise the model. 

    model.addExitFunction(getMeanMessageCount);


#if defined(CIRCLES_VALIDATION) && CIRCLES_VALIDATION
    {   // Attach init/step/exit functions and exit condition
        model.addStepFunction(Validation);
    }
#endif  // CIRCLES_VALIDATION

    {   // Layer #1
        flamegpu::LayerDescription &layer = model.newLayer();
        layer.addAgentFunction("Circle", "output_message");
    }
    {   // Layer #2
        flamegpu::LayerDescription &layer = model.newLayer();
        layer.addAgentFunction("Circle", "move");
    }

    // Create the simulation object
    flamegpu::CUDASimulation simulation(model);

    // Set config configuraiton properties 
    simulation.SimulationConfig().timing = false;
    simulation.SimulationConfig().verbose = false;
    simulation.SimulationConfig().random_seed = runInputs.SEED;
    simulation.SimulationConfig().steps = runInputs.STEPS;
    simulation.CUDAConfig().device_id = runInputs.CUDA_DEVICE;

    // Generate the initial population
    std::mt19937_64 rng(runInputs.SEED);
    std::uniform_real_distribution<float> dist(ENV_MIN, ENV_MAX);
    flamegpu::AgentVector population(model.Agent("Circle"), runInputs.AGENT_COUNT);
    for (unsigned int i = 0; i < runInputs.AGENT_COUNT; i++) {
        flamegpu::AgentVector::Agent instance = population[i];
        instance.setVariable<int>("id", i);
        instance.setVariable<float>("x", dist(rng));
        instance.setVariable<float>("y", dist(rng));
        instance.setVariable<float>("z", dist(rng));
    }

    // Set the population for the simulation.
    simulation.setPopulationData(population);

    // Execute 
    simulation.simulate();

    // Store timing information for later use.
    runOutputs.s_rtc = simulation.getElapsedTimeRTCInitialisation();
    runOutputs.s_simulation = simulation.getElapsedTimeSimulation();
    runOutputs.s_init = simulation.getElapsedTimeInitFunctions();
    runOutputs.s_exit = simulation.getElapsedTimeExitFunctions();
    
    std::vector<double> s_steps = simulation.getElapsedTimeSteps();
    runOutputs.s_per_step = std::make_shared<std::vector<double>>(std::vector<double>(s_steps.begin(), s_steps.end()));
    runOutputs.s_stepMean = std::accumulate(s_steps.begin(), s_steps.end(), 0.f) / (double)simulation.getStepCounter();
    runOutputs.mean_messageCount = meanMessageCount;
}